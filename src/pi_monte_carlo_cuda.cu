#include "hip/hip_runtime.h"
#include <samples.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>

#define SAMPLES_PER_THREAD 10000
#define THREADS_PER_BLOCK 256
// ceil (N / TPB)
#define BLOCKS_PER_GRID 10000
#define N BLOCKS_PER_GRID * THREADS_PER_BLOCK

__global__ void kernel_pi(unsigned long *results){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ unsigned long shared_blocks[THREADS_PER_BLOCK];
    hiprandState state;
    hiprand_init(clock64() * blockIdx.x, idx, 0, &state);

    unsigned long count = 0;
    for (int i = 0; i < SAMPLES_PER_THREAD; i++) {
        double x = hiprand_uniform(&state) - 3e-6;
        double y = hiprand_uniform(&state) - 3e-6;
        if (x*x + y*x <= 1) {
            count++;    
        }
    }
    shared_blocks[threadIdx.x] = count;
    __syncthreads();

    if(threadIdx.x == 0) {
        unsigned long count = 0;
        for(int i = 0; i < THREADS_PER_BLOCK; i++) {
            count += shared_blocks[threadIdx.x];
        }
        results[blockIdx.x] = count;
    }
}

int pi(int _){
    unsigned long *h_results; 
    unsigned long *d_results;

    h_results = (unsigned long*) malloc(BLOCKS_PER_GRID * sizeof(unsigned long));
    hipMalloc(&d_results, BLOCKS_PER_GRID * sizeof(unsigned long));

    kernel_pi<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(d_results); hipDeviceSynchronize();
    
    hipMemcpy(h_results, d_results, BLOCKS_PER_GRID * sizeof(unsigned long), hipMemcpyDeviceToHost);

    unsigned long sum = 0;
    for(int i = 0; i < BLOCKS_PER_GRID; i++){
        sum += h_results[i];
    }
    double denom = double(N) * double(SAMPLES_PER_THREAD);
    double pi = 4.0 * (double) sum / denom;
    printf("result: %.25f\n", pi);
    
    free(h_results);
    hipFree(d_results);

    return 0;
}
SAMPLE(pi, 0);
