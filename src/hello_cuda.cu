#include "hip/hip_runtime.h"
#include <samples.h>
#include <stdio.h>

__global__ void kernel()
{
    printf("Hello from the GPU! Thread ID: %d\n", threadIdx.x);
}

int hello(int _){
    kernel<<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}
SAMPLE(hello, 0);
