#include "hip/hip_runtime.h"
#include <samples.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 1000000
#define THREADS_PER_BLOCK 256
// ceil (N / TPB)
#define BLOCKS_PER_GRID (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK 

__global__ void vadd(float *A, float *B, float *C){
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix >= N) {
        return;
    }
    C[ix] = A[ix] + B[ix];
}

int vector_add(int _){

    // host
    float *h_A, *h_B, *h_C; 
    float *d_A, *d_B, *d_C;  // device 
    srand(time(NULL));

    h_A = (float*) malloc(N * sizeof(float));
    h_B = (float*) malloc(N * sizeof(float));
    h_C = (float*) malloc(N * sizeof(float));

    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    /* generate data */
    for (int i = 0; i < N; i++){
        h_A[i] = (float) rand() / (float) RAND_MAX;
        h_B[i] = (float) rand() / (float) RAND_MAX;
    }

    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    vadd<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(d_A, d_B, d_C); hipDeviceSynchronize();
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%f ", h_A[i]);
    }
    printf("\n");

    for (int i = 0; i < 10; i++) {
        printf("%f ", h_B[i]);
    }
    printf("\n");

    for (int i = 0; i < 10; i++) {
        printf("%f ", h_C[i]);
    }
    printf("\n");
    
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
SAMPLE(vector_add, 0);
